#include "hip/hip_runtime.h"

/*
Copyright (c) 2012, MAURO BIANCO, UGO VARETTO, SWISS NATIONAL SUPERCOMPUTING CENTRE (CSCS)
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of the Swiss National Supercomputing Centre (CSCS) nor the
      names of its contributors may be used to endorse or promote products
      derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL MAURO BIANCO, UGO VARETTO, OR 
SWISS NATIONAL SUPERCOMPUTING CENTRE (CSCS), BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/


#include <GCL.h>
#include <iostream>
#include <sstream>
#include <fstream>
std::ostream *filep;

#include <halo_exchange.h>
#include <string>
#include <stdlib.h>
#include <utils/layout_map.h>
#include <utils/boollist.h>
#include <sys/time.h>

#include "triplet.h"

int pid;
int nprocs;
MPI_Comm CartComm;
int dims[3] = {0,0,0};
int coords[3]={0,0,0};

int datalen, tloop;
int tot_it;
double *thedata;
double* compdata_g;


__global__ void kernel_ex(double *a, int N, int _tloop)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx<N) 
      for (int i=0; i < _tloop; ++i)
          a[idx] += std::pow(a[idx],01.0056);
}

void prepare_to_do_something() {
    hipMalloc((void**)&compdata_g, sizeof(double)*datalen);
    hipMemcpy(thedata, compdata_g, sizeof(double)*datalen, hipMemcpyDeviceToHost);
}

hipStream_t c_stream ;


void do_something() {
  dim3 grid_size, block_size;
  block_size.x = 32;
  block_size.y = 4;
  grid_size.x = datalen / (block_size.x*block_size.y) +1;
  
  kernel_ex <<< grid_size, block_size, 0, c_stream >>> (compdata_g, datalen, tloop); 
  //  hipDeviceSynchronize();

  //  hipStreamDestroy ( c_stream );
}

struct timeval start_tv;
struct timeval stop1_tv;
struct timeval stop2_tv;
struct timeval stop3_tv;
double lapse_time1;
double lapse_time2;
double lapse_time3;
double lapse_time4;

#ifndef PACKING_TYPE
#define PACKING_TYPE GCL::version_manual
#endif

#define B_ADD 1
#define C_ADD 2

typedef GCL::gcl_gpu arch_type;

template <typename T, typename lmap>
struct array {
  T *ptr;
 int n,m,l;

  array(T* _p, int _n, int _m, int _l)
    : ptr(_p)
    , n(lmap::template find<2>(_n,_m,_l))
    , m(lmap::template find<1>(_n,_m,_l))
    , l(lmap::template find<0>(_n,_m,_l))  
  {}

  T &operator()(int i, int j, int k) {
    // a[(DIM1+2*H)*(DIM2+2*H)*kk+ii*(DIM2+2*H)+jj]
    return ptr[l*m*lmap::template find<2>(i,j,k)+
               l*lmap::template find<1>(i,j,k)+
               lmap::template find<0>(i,j,k)];
  }

  T const &operator()(int i, int j, int k) const {
    return ptr[l*m*lmap::template find<2>(i,j,k)+
               l*lmap::template find<1>(i,j,k)+
               lmap::template find<0>(i,j,k)];
  }

  operator void*() const {return reinterpret_cast<void*>(ptr);}
  operator T*() const {return ptr;}
};

/** \file Example of use of halo_exchange pattern for regular
    grids. The comments in the code aim at highlight the process of
    instantiating and running a halo exchange pattern.
*/

inline int modulus(int __i, int __j) {
  return (((((__i%__j)<0)?(__j+__i%__j):(__i%__j))));
}

/* Just and utility to print values
 */
template <typename array_t>
void printbuff(std::ostream &file, array_t const & a, int d1, int d2, int d3) {
  if (d1<=7 && d2<=7 && d3<=7) {
    file << "------------\n";
    for (int kk=0; kk<d3; ++kk) {
      file << "|";
      for (int jj=0; jj<d2; ++jj) {
        for (int ii=0; ii<d1; ++ii) {
          file << a(ii,jj,kk);
        }
        file << "|\n";
      }
      file << "\n\n";
    }
    file << "------------\n\n";
  }
}


template <typename ST, int I1, int I2, int I3, bool per0, bool per1, bool per2>
void run(ST & file, int DIM1, int DIM2, int DIM3, int H1, int H2, int H3, triple_t<USE_DOUBLE> *_a, triple_t<USE_DOUBLE> *_b, triple_t<USE_DOUBLE> *_c) {

  typedef GCL::layout_map<I1,I2,I3> layoutmap;
  
  array<triple_t<USE_DOUBLE>, layoutmap > a(_a, (DIM1+2*H1),(DIM2+2*H2),(DIM3+2*H3));
  array<triple_t<USE_DOUBLE>, layoutmap > b(_b, (DIM1+2*H1),(DIM2+2*H2),(DIM3+2*H3));
  array<triple_t<USE_DOUBLE>, layoutmap > c(_c, (DIM1+2*H1),(DIM2+2*H2),(DIM3+2*H3));

  /* Just an initialization */
  for (int ii=0; ii<DIM1+2*H1; ++ii)
    for (int jj=0; jj<DIM2+2*H2; ++jj) {
      for (int kk=0; kk<DIM3+2*H3; ++kk) {
        a(ii,jj,kk) = triple_t<USE_DOUBLE>();
        b(ii,jj,kk) = triple_t<USE_DOUBLE>();                                      
        c(ii,jj,kk) = triple_t<USE_DOUBLE>();
      }
    }
//   a(0,0,0) = triple_t<USE_DOUBLE>(3000+GCL::PID, 4000+GCL::PID, 5000+GCL::PID);
//   b(0,0,0) = triple_t<USE_DOUBLE>(3010+GCL::PID, 4010+GCL::PID, 5010+GCL::PID);
//   c(0,0,0) = triple_t<USE_DOUBLE>(3020+GCL::PID, 4020+GCL::PID, 5020+GCL::PID);


  /* The pattern type is defined with the layouts, data types and
     number of dimensions.

     The logical assumption done in the program is that 'i' is the
     first dimension (rows), 'j' is the second, and 'k' is the
     third. The first layout states that 'i' is the second dimension
     in order of strides, while 'j' is the first and 'k' is the third
     (just by looking at the initialization loops this shoule be
     clear).

     The second layout states that the first dimension in data ('i')
     identify also the first dimension in the communicator. Logically,
     moving on 'i' dimension from processot (p,q,r) will lead you
     logically to processor (p+1,q,r). The other dimensions goes as
     the others.
   */
  typedef GCL::halo_exchange_generic<GCL::layout_map<0,1,2>, 3, arch_type, PACKING_TYPE > pattern_type;


  /* The pattern is now instantiated with the periodicities and the
     communicator. The periodicity of the communicator is
     irrelevant. Setting it to be periodic is the best choice, then
     GCL can deal with any periodicity easily.
  */
  pattern_type he(typename pattern_type::grid_type::period_type(per0, per1, per2), CartComm);


  GCL::array<GCL::halo_descriptor,3> halo_dsc;
  halo_dsc[0] = GCL::halo_descriptor(H1, H1, H1, DIM1+H1-1, DIM1+2*H1);
  halo_dsc[1] = GCL::halo_descriptor(H2, H2, H2, DIM2+H2-1, DIM2+2*H2);
  halo_dsc[2] = GCL::halo_descriptor(H3, H3, H3, DIM3+H3-1, DIM3+2*H3);

  GCL::field_on_the_fly<triple_t<USE_DOUBLE>::data_type, layoutmap, pattern_type::traits> field1(reinterpret_cast<triple_t<USE_DOUBLE>::data_type*>(a.ptr), halo_dsc);
  GCL::field_on_the_fly<triple_t<USE_DOUBLE>::data_type, layoutmap, pattern_type::traits> field2(reinterpret_cast<triple_t<USE_DOUBLE>::data_type*>(b.ptr), halo_dsc);
  GCL::field_on_the_fly<triple_t<USE_DOUBLE>::data_type, layoutmap, pattern_type::traits> field3(reinterpret_cast<triple_t<USE_DOUBLE>::data_type*>(c.ptr), halo_dsc);

  /* Pattern is set up. This must be done only once per pattern. The
     parameter must me greater or equal to the largest number of
     arrays updated in a single step.
  */
  //he.setup(100, halo_dsc, sizeof(double));
  he.setup(3, GCL::field_on_the_fly<int,layoutmap, pattern_type::traits>(NULL,halo_dsc), sizeof(triple_t<USE_DOUBLE>)); // Estimates the size

  file << "Proc: (" << coords[0] << ", " << coords[1] << ", " << coords[2] << ")\n";


  /* Data is initialized in the inner region of size DIM1xDIM2
   */
  for (int ii=H1; ii<DIM1+H1; ++ii)
    for (int jj=H2; jj<DIM2+H2; ++jj) 
      for (int kk=H3; kk<DIM3+H3; ++kk) {
        a(ii,jj,kk) = //(100*(pid))+
          triple_t<USE_DOUBLE>(ii-H1+(DIM1)*coords[0],
                   jj-H2+(DIM2)*coords[1],
                   kk-H3+(DIM3)*coords[2]);
          b(ii,jj,kk) = //(200*(pid))+ 
          triple_t<USE_DOUBLE>(ii-H1+(DIM1)*coords[0]+B_ADD,
                   jj-H2+(DIM2)*coords[1]+B_ADD,
                   kk-H3+(DIM3)*coords[2]+B_ADD);
          c(ii,jj,kk) = //300*(pid))+
          triple_t<USE_DOUBLE>(ii-H1+(DIM1)*coords[0]+C_ADD,
                   jj-H2+(DIM2)*coords[1]+C_ADD,
                   kk-H3+(DIM3)*coords[2]+C_ADD);
      }

  file << "A \n";
  printbuff(file,a, DIM1+H1+H1, DIM2+H2+H2, DIM3+H3+H3);
  file << "B \n";
  printbuff(file,b, DIM1+H1+H1, DIM2+H2+H2, DIM3+H3+H3);
  file << "C \n";
  printbuff(file,c, DIM1+H1+H1, DIM2+H2+H2, DIM3+H3+H3);
  file.flush();

  file << "GPU GPU GPU GPU GPU GPU GPU GPU GPU GPU GPU GPU GPU GPU GPU GPU \n";

  triple_t<USE_DOUBLE>* gpu_a = 0;
  triple_t<USE_DOUBLE>* gpu_b = 0;
  triple_t<USE_DOUBLE>* gpu_c = 0;
  hipError_t status;
  status = hipMalloc( &gpu_a, (DIM1+2*H1)*(DIM2+2*H2)*(DIM3+2*H3)*sizeof(triple_t<USE_DOUBLE>));
  if( !checkCudaStatus( status ) ) return;
  status = hipMalloc( &gpu_b, (DIM1+2*H1)*(DIM2+2*H2)*(DIM3+2*H3)*sizeof(triple_t<USE_DOUBLE>));
  if( !checkCudaStatus( status ) ) return;
  status = hipMalloc( &gpu_c, (DIM1+2*H1)*(DIM2+2*H2)*(DIM3+2*H3)*sizeof(triple_t<USE_DOUBLE>));
  if( !checkCudaStatus( status ) ) return;

  status = hipMemcpy( gpu_a, a.ptr,
                       (DIM1+2*H1)*(DIM2+2*H2)*(DIM3+2*H3)*sizeof(triple_t<USE_DOUBLE>), 
                       hipMemcpyHostToDevice );
  if( !checkCudaStatus( status ) ) return;

  status = hipMemcpy( gpu_b, b.ptr,
                       (DIM1+2*H1)*(DIM2+2*H2)*(DIM3+2*H3)*sizeof(triple_t<USE_DOUBLE>), 
                       hipMemcpyHostToDevice );
  if( !checkCudaStatus( status ) ) return;

  status = hipMemcpy( gpu_c, c.ptr,
                       (DIM1+2*H1)*(DIM2+2*H2)*(DIM3+2*H3)*sizeof(triple_t<USE_DOUBLE>), 
                       hipMemcpyHostToDevice );
  if( !checkCudaStatus( status ) ) return;


  GCL::field_on_the_fly<triple_t<USE_DOUBLE>::data_type, layoutmap, pattern_type::traits> field1_gpu(reinterpret_cast<triple_t<USE_DOUBLE>::data_type*>(gpu_a), halo_dsc);
  GCL::field_on_the_fly<triple_t<USE_DOUBLE>::data_type, layoutmap, pattern_type::traits> field2_gpu(reinterpret_cast<triple_t<USE_DOUBLE>::data_type*>(gpu_b), halo_dsc);
  GCL::field_on_the_fly<triple_t<USE_DOUBLE>::data_type, layoutmap, pattern_type::traits> field3_gpu(reinterpret_cast<triple_t<USE_DOUBLE>::data_type*>(gpu_c), halo_dsc);
  std::vector<GCL::field_on_the_fly<triple_t<USE_DOUBLE>::data_type, layoutmap, pattern_type::traits> > vect(3);


  gettimeofday(&start_tv, NULL);

  MPI_Barrier(MPI_COMM_WORLD);

  for (int n_it = 0; n_it < tot_it; ++n_it) {
      he.post_receives();
      he.pack(field1_gpu, field2_gpu, field3_gpu);

      //MPI_Barrier(MPI_COMM_WORLD);
      he.do_sends();

      do_something();

      he.wait();

      he.unpack(field1_gpu, field2_gpu, field3_gpu);
      //      MPI_Barrier(MPI_COMM_WORLD);
  }


  hipDeviceSynchronize();
  gettimeofday(&stop3_tv, NULL);

  lapse_time4 = ((static_cast<double>(stop3_tv.tv_sec)+1/1000000.0*static_cast<double>(stop3_tv.tv_usec)) - (static_cast<double>(start_tv.tv_sec)+1/1000000.0*static_cast<double>(start_tv.tv_usec))) * 1000.0;

  MPI_Barrier(MPI_COMM_WORLD);
  file << "TIME PACK: " << lapse_time1 << std::endl;
  file << "TIME EXCH: " << lapse_time2 << std::endl;
  file << "TIME UNPK: " << lapse_time3 << std::endl;
  file << "TIME ALL : " << lapse_time1+lapse_time2+lapse_time3 << std::endl;
  file << "TIME TOT : " << lapse_time4 << std::endl;

  status = hipMemcpy( a.ptr, gpu_a,
                       (DIM1+2*H1)*(DIM2+2*H2)*(DIM3+2*H3)*sizeof(triple_t<USE_DOUBLE>), 
                       hipMemcpyDeviceToHost );
  if( !checkCudaStatus( status ) ) return;

  status = hipMemcpy( b.ptr, gpu_b,
                       (DIM1+2*H1)*(DIM2+2*H2)*(DIM3+2*H3)*sizeof(triple_t<USE_DOUBLE>), 
                       hipMemcpyDeviceToHost );
  if( !checkCudaStatus( status ) ) return;

  status = hipMemcpy( c.ptr, gpu_c,
                       (DIM1+2*H1)*(DIM2+2*H2)*(DIM3+2*H3)*sizeof(triple_t<USE_DOUBLE>), 
                       hipMemcpyDeviceToHost );
  if( !checkCudaStatus( status ) ) return;

  status = hipFree( gpu_a );
  if( !checkCudaStatus( status ) ) return;
  status = hipFree( gpu_b );
  if( !checkCudaStatus( status ) ) return;
  status = hipFree( gpu_c );
  if( !checkCudaStatus( status ) ) return;

  file << "\n********************************************************************************\n";

  file << "A \n";
  printbuff(file,a, DIM1+H1+H1, DIM2+H2+H2, DIM3+H3+H3);
  file << "B \n";
  printbuff(file,b, DIM1+H1+H1, DIM2+H2+H2, DIM3+H3+H3);
  file << "C \n";
  printbuff(file,c, DIM1+H1+H1, DIM2+H2+H2, DIM3+H3+H3);
  file.flush();
  int passed = true;


  /* Checking the data arrived correctly in the whole region
   */
  for (int ii=0; ii<DIM1+2*H1; ++ii)
    for (int jj=0; jj<DIM2+2*H2; ++jj)
      for (int kk=0; kk<DIM3+2*H3; ++kk) {

        triple_t<USE_DOUBLE> ta;
        triple_t<USE_DOUBLE> tb;
        triple_t<USE_DOUBLE> tc;
        int tax, tay, taz;
        int tbx, tby, tbz;
        int tcx, tcy, tcz;

        tax = modulus(ii-H1+(DIM1)*coords[0], DIM1*dims[0]);
        tbx = modulus(ii-H1+(DIM1)*coords[0], DIM1*dims[0])+B_ADD;
        tcx = modulus(ii-H1+(DIM1)*coords[0], DIM1*dims[0])+C_ADD;

        tay = modulus(jj-H2+(DIM2)*coords[1], DIM2*dims[1]);
        tby = modulus(jj-H2+(DIM2)*coords[1], DIM2*dims[1])+B_ADD;
        tcy = modulus(jj-H2+(DIM2)*coords[1], DIM2*dims[1])+C_ADD;

        taz = modulus(kk-H3+(DIM3)*coords[2], DIM3*dims[2]);
        tbz = modulus(kk-H3+(DIM3)*coords[2], DIM3*dims[2])+B_ADD;
        tcz = modulus(kk-H3+(DIM3)*coords[2], DIM3*dims[2])+C_ADD;

        if (!per0) {
          if ( ((coords[0]==0) && (ii<H1)) || 
               ((coords[0] == dims[0]-1) && (ii >= DIM1+H1)) ) {
            tax=triple_t<USE_DOUBLE>().x();
            tbx=triple_t<USE_DOUBLE>().x();
            tcx=triple_t<USE_DOUBLE>().x();
          }
        }

        if (!per1) {
          if ( ((coords[1]==0) && (jj<H2)) || 
               ((coords[1] == dims[1]-1) && (jj >= DIM2+H2)) ) {
            tay=triple_t<USE_DOUBLE>().y();
            tby=triple_t<USE_DOUBLE>().y();
            tcy=triple_t<USE_DOUBLE>().y();
          }
        }

        if (!per2) {
          if ( ((coords[2]==0) && (kk<H3)) || 
               ((coords[2] == dims[2]-1) && (kk >= DIM3+H3)) ) {
            taz=triple_t<USE_DOUBLE>().z();
            tbz=triple_t<USE_DOUBLE>().z();
            tcz=triple_t<USE_DOUBLE>().z();
          }
        }

        ta = triple_t<USE_DOUBLE>(tax, tay, taz).floor();
        tb = triple_t<USE_DOUBLE>(tbx, tby, tbz).floor();
        tc = triple_t<USE_DOUBLE>(tcx, tcy, tcz).floor();

        if (a(ii,jj,kk) != ta) {
          passed = false;
          file << ii << ", " << jj << ", " << kk << " values found != expct: " 
               << "a " << a(ii,jj,kk) << " != " 
               << ta
               << "\n";
        }

        if (b(ii,jj,kk) != tb) {
          passed = false;
          file << ii << ", " << jj << ", " << kk << " values found != expct: " 
               << "b " << b(ii,jj,kk) << " != " 
               << tb
               << "\n";
        }

        if (c(ii,jj,kk) != tc) {
          passed = false;
          file << ii << ", " << jj << ", " << kk << " values found != expct: " 
               << "c " << c(ii,jj,kk) << " != " 
               << tc
               << "\n";
        }
      }

  if (passed)
    file << "RESULT: PASSED!\n";
  else
    file << "RESULT: FAILED!\n";
}

#ifdef _GCL_GPU_
/* device_binding added by Devendar Bureddy, OSU */

void
device_binding ()
{

  int local_rank=0/*, num_local_procs*/;
  int dev_count, use_dev_count, my_dev_id;
  char *str;

  if ((str = getenv ("MV2_COMM_WORLD_LOCAL_RANK")) != NULL)
    {
      local_rank = atoi (str);
      printf ("MV2_COMM_WORLD_LOCAL_RANK %s\n", str);
    }

  if ((str = getenv ("MPISPAWN_LOCAL_NPROCS")) != NULL)
    {
      //num_local_procs = atoi (str);
      printf ("MPISPAWN_LOCAL_NPROCS %s\n", str);
    }

  hipGetDeviceCount (&dev_count);
  if ((str = getenv ("NUM_GPU_DEVICES")) != NULL)
    {
      use_dev_count = atoi (str);
      printf ("NUM_GPU_DEVICES %s\n", str);
    }
  else
    {
      use_dev_count = dev_count;
    }

  my_dev_id = local_rank % use_dev_count;
  printf ("local rank = %d dev id = %d\n", local_rank, my_dev_id);
  hipSetDevice (my_dev_id);
}
#endif

int main(int argc, char** argv) {

#ifdef _GCL_GPU_
  device_binding();
#endif

  /* this example is based on MPI Cart Communicators, so we need to
  initialize MPI. This can be done by GCL automatically
  */
  GCL::GCL_Init(argc, argv);


  /* Now let us initialize GCL itself. If MPI is not initialized at
     this point, it will initialize it
   */
  GCL::GCL_Init(argc, argv);

  /* Here we compute the computing gris as in many applications
   */
  MPI_Comm_rank(MPI_COMM_WORLD, &pid);
  MPI_Comm_size(MPI_COMM_WORLD, &nprocs);

  if (argc != 9) {
      if (pid==0) {
          std::cout << "Usage: " << argv[0] << " dim1 dim2 dim3 h datalen tloop tot_it suffix\n"
                    << "where:\n"
                    << "dim1 dim2 dim3 are the sizes of the 3D tiles in each process.\n"
                    << "h is the halo width around the above dimensions.\n"
                    << "datalen is the amount of data to allocated (on GPU) for performing the work to be overlapped.\n"
                    << "        There will be a kernel launch with this many threads.\n"
                    << " tloop is the number of iterations each GPU thread will execute\n"
                    << "tot_it is the number of halo exchanges to be executed before measuring time\n"
                    << "suffix is a string to be appended to output files to be able to run multiple jobs at the same time\n"
                    << std::endl;
          return 1;
      }
  }

  std::cout << pid << " " << nprocs << "\n";

  std::stringstream ss;
  ss << pid;

  std::string suffix(argv[8]);

  std::string filename = "out" + ss.str() + suffix + ".txt" ;

  std::cout << filename << std::endl;
  std::ofstream file(filename.c_str());

  filep = &file;

  file << pid << "  " << nprocs << "\n";

  MPI_Dims_create(nprocs, 3, dims);
  int period[3] = {1, 1, 1};

  file << "@" << pid << "@ MPI GRID SIZE " << dims[0] << " - " << dims[1] << " - " << dims[2] << "\n";
 
  MPI_Cart_create(MPI_COMM_WORLD, 3, dims, period, false, &CartComm);

  MPI_Cart_get(CartComm, 3, dims, period, coords);


  /* Each process will hold a tile of size
     (DIM1+2*H)x(DIM2+2*H)x(DIM3+2*H). The DIM1xDIM2xDIM3 area inside
     the H width border is the inner region of an hypothetical stencil
     computation whise halo width is H.
   */
  int DIM1=atoi(argv[1]);
  int DIM2=atoi(argv[2]);
  int DIM3=atoi(argv[3]);
  int H1  =atoi(argv[4]);
  int H2=H1;
  int H3=H1;
  datalen =atoi(argv[5]);
  if (datalen)
      thedata = new double[datalen];
  tloop =atoi(argv[6]);
  tot_it  =atoi(argv[7]);

  hipStreamCreate ( &c_stream );

  /* This example will exchange 3 data arrays at the same time with
     different values.
   */
  triple_t<USE_DOUBLE> *_a = new triple_t<USE_DOUBLE>[(DIM1+2*H1)*(DIM2+2*H2)*(DIM3+2*H3)];
  triple_t<USE_DOUBLE> *_b = new triple_t<USE_DOUBLE>[(DIM1+2*H1)*(DIM2+2*H2)*(DIM3+2*H3)];
  triple_t<USE_DOUBLE> *_c = new triple_t<USE_DOUBLE>[(DIM1+2*H1)*(DIM2+2*H2)*(DIM3+2*H3)];

  prepare_to_do_something();

  file << "Permutation 0,1,2\n";

#ifndef BENCH
#define BENCH 5
#endif

  for (int i=0; i<BENCH; ++i) {
    file << "run<std::ostream, 0,1,2, true, true, true>(file, DIM1, DIM2, DIM3, H1, H2, H3, _a, _b, _c)\n";
    run<std::ostream, 0,1,2, true, true, true>(file, DIM1, DIM2, DIM3, H1, H2, H3, _a, _b, _c);
    file.flush();
  }


  MPI_Barrier(MPI_COMM_WORLD);
  MPI_Finalize();
  return 0;
}
